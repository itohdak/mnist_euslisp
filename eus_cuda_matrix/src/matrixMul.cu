#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication as described in Chapter 3
 * of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 */
template <int BLOCK_SIZE> __global__ void
matrixMulCUDA(double *C, double *A, double *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    double Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int matrixMultiply(int row_A, int col_A, int col_B, double *h_A, double *h_B, double *h_C)
{
    // int devID;
    // hipDeviceProp_t deviceProp;
    // hipGetDevice(&devID);
    // hipGetDeviceProperties(&deviceProp, devID);
    // printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

    const int block_size = 32;
    dim3 dimsA(col_A, row_A, 1);
    dim3 dimsB(col_B, col_A, 1);
    dim3 dimsC(col_B, row_A, 1);
    unsigned int mem_size_A = sizeof(double) * dimsA.x * dimsA.y;
    unsigned int mem_size_B = sizeof(double) * dimsB.x * dimsB.y;
    unsigned int mem_size_C = sizeof(double) * dimsC.x * dimsC.y;

    // Allocate device memory
    double *d_A, *d_B, *d_C;
    hipMalloc((void **) &d_A, mem_size_A);
    hipMalloc((void **) &d_B, mem_size_B);
    hipMalloc((void **) &d_C, mem_size_C);

    // Copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Calculate
    matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    // hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    // Clean up memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("matrixMultiply is finished successfully.\n");

    return 0;
}

extern "C" {
  int call_matrixMultiply(int row_A, int col_A, int col_B, double *h_A, double *h_B, double *h_C)
  {
    return matrixMultiply(row_A, col_A, col_B, h_A, h_B, h_C);
  }
}
